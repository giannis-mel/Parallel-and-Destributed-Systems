#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
 
#include <hip/hip_runtime_api.h>
#include <string.h>

float s = 1;
#define points 600
#define dimensions 2



//gia compile : nvcc nonshared.cu -o nonshared -O3 -lm 
//gia run: ./nonshared 0.5









//sinartisi pou epistrefei tin apostasi 2 simiwn
__device__ float distanceFunction(float point1[2],float point2[2]){

  float distance;
  int i;
  distance=0;

  for(i=0;i<dimensions;i++){
    distance+=(point1[i]-point2[i])*(point1[i]-point2[i]);
  }

  distance = sqrt(distance);
  return distance;
}

//sinartisi pou epistrefei to varos
__device__ float kernelFunction(float pointYk[2],float arrayXi[2]){

  float s=1;
  float returnWeight=0;
  float distance=distanceFunction(pointYk,arrayXi);
  distance*=distance;

  returnWeight=exp(-((distance)/(2*(s*s))));
  return returnWeight;
}

//epistrefei tin apostasi pou kineitai to moved
__device__ float movedDistance(float moved[2]){
  float distance;
  int i;
  distance=0;

  for(i=0;i<dimensions;i++){
    distance+=(moved[i])*(moved[i]);
  }

  distance = sqrt(distance);
  return distance;
}













__global__ void shiftingFunction(float *Ykplus1, float *Yk, float *X, float e){
	
	
	
	int index = blockIdx.x *blockDim.x +  threadIdx.x;
	//printf("index is %d\n",index);
		
 int i,j;

  /*//tipwnei tis 2 diastaseis kathe simiou (elegxos)
  for(i=0;i<dimensions;i++){
    printf("%f ",pointYk[i]);
  }
  printf("\n");
*/


  int s=1;
  
  float arithmitis[dimensions];
  for(i=0;i<dimensions;i++) { arithmitis[i]=0;}
  float paronomastis=0;
  float weightFromGaussian=0;
  float check,S=s*s;

  float Ypoint[2],Xpoint[2],moved[2];
  


  
  moved[0]=9;moved[1]=9;
  
  
  while(movedDistance(moved)>=e){
	  
	    //auto ine to simio pou kathorizete apo to index
	  Ypoint[0]=Yk[index*2];
      Ypoint[1]=Yk[index*2+1];
	  
	  paronomastis=0;
      weightFromGaussian=0;
	  for(i=0;i<dimensions;i++) { arithmitis[i]=0;}
	  
	//ipologizw gia kathe simio tou X an tha mpei sti sinartisi
	for(i=0;i<points;i++){
		
		
		Xpoint[0]=X[i*2];
		Xpoint[1]=X[i*2+1];
		
		check=distanceFunction(Ypoint,Xpoint);
		
		//printf("distance(%d)=%f and s^2=%f\n",i,check,S);
		
		if(check<=S && check>0) 
		{
		weightFromGaussian=kernelFunction(Ypoint,Xpoint);
		for(j=0;j<dimensions;j++){
			arithmitis[j]+= weightFromGaussian*X[i*dimensions+j];
		}
		paronomastis+=weightFromGaussian;
		}
		else { }
	}

	for(j=0;j<dimensions;j++){
		Ykplus1[index*2+j]=arithmitis[j]/paronomastis;
	}


	for(j=0;j<dimensions;j++){
		moved[j]=Ykplus1[index*2+j]-Yk[index*2+j];
		Yk[index*2+j]=Ykplus1[index*2+j];
	}


  }





  
}

















int main(int argc, char **argv){

  //anoigma tou arxeiou gia diavasma
  FILE *myFile = fopen("eisodos.txt", "r");
  //elegxos an den anikse
  if(myFile==0){
    fprintf(stderr,"Failed to open eisodos.txt.\n");
    return 1;
  }

  
  /*
  if(freopen("stdout.txt","w",stdout)==0) {
	fprintf(stderr,"Failed to write in stdout.txt.\n");
    return 1;
  }*/
  
  
  //oi pinakes tha einai 2d->1d
  
  //pinakes tou host
  float *arrayStatic, *arrayYk /*,*arrayYkplus1*/;
  //pinakes tou device
  float *deviceArrayStatic, *deviceArrayYk, *deviceArrayYkplus1;
  
  
  
  //megethos pinakwn
  int nBytes=points*dimensions*sizeof(float);
  
  //malloc gia pinakes tou host
  arrayStatic=(float *)malloc(nBytes);
  arrayYk=(float *)malloc(nBytes);
  //arrayYkplus1=(float *)malloc(nBytes);
  
  //malloc gia pinakes tou device
  hipMalloc((void **)&deviceArrayStatic,nBytes);
  hipMalloc((void **)&deviceArrayYk,nBytes);
  hipMalloc((void **)&deviceArrayYkplus1,nBytes);
  
  
  int i,j,scanfReturn;
  float myvar;
  
  //diavasma tou pinaka apo to arxeio dataset.txt
  for(i=0;i<points;i++){
	for(j=0;j<dimensions;j++){
      //den diavazei ton xaraktira meta to float eite ine , eite ine \n
      scanfReturn=fscanf(myFile,"%f%*c",&myvar);

      //elegxos an i scanf litourgise swsta
      if(scanfReturn==1){/*ola kala*/}
      else{ printf("error in %d,%d. \n",i,j); return 1;}

      //ekxwrisi timwn ston pinaka arrayStatic
      arrayStatic[i*dimensions+j]=myvar;
      //stin arxi ta simia tou arrayYk einai idia me ta sima tou arrayStatic
      arrayYk[i*dimensions+j]=myvar;
    }
  }

  printf("Finished reading from file\n");
  fclose(myFile);
  
  //metafora apo tous pinakes tou host stous pinakes tou device
  hipMemcpy(deviceArrayStatic,arrayStatic,nBytes,hipMemcpyHostToDevice);
  hipMemcpy(deviceArrayYk,arrayYk,nBytes,hipMemcpyHostToDevice);
  
  
  struct timeval startwtime, endwtime;
  float seq_time;
  //xronos arxis
  gettimeofday (&startwtime, NULL);
  
  
  //pairnw stathera kai tin vazw sto e
  const char* text = argv[1];
  float e=atof(text); 
  
  
  printf("e=%f \nBefore shiftingFunction\n",e);

  
  //edw ginetai o algorithmos MeanShift
  shiftingFunction<<< 6, 100 >>>(deviceArrayYkplus1,deviceArrayYk,deviceArrayStatic,e);
  hipDeviceSynchronize();
	
 
	
  //telos metrisis tou xronou 
  gettimeofday (&endwtime, NULL); 
  seq_time = (float)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
  printf("After shiftingFunction\nclock time = %f\n", 1000*seq_time);
  
  hipMemcpy(arrayYk,deviceArrayYk,nBytes,hipMemcpyDeviceToHost);
  //apo edw kai pera exw ston arrayYk ta apotelesmata
  
  
    FILE *f = fopen("output.txt", "w");
    if (f == NULL){
      printf("Error opening file!\n");
      exit(1);
    }

    for(i=0;i<points;i++){
      for(j=0;j<dimensions;j++){
        fprintf(f,"%f ",arrayYk[i*dimensions+j]);
      }
      fprintf(f,"\n");
    }


	

    fclose(f);
	
	
	//EDW NA KANW TON ELEGXO ORTHOTITAS TOU ALGORITHMOU
    
	FILE *f2=fopen("eksodos.txt", "r");
	
	
	float myvar2;
	int errors=0;
	
		for(i=0;i<points;i++){
		for(j=0;j<dimensions;j++){
			
			//den diavazei ton xaraktira meta to float eite ine , eite ine \n
			scanfReturn=fscanf(f2,"%f%*c",&myvar2);
			//elegxos an i scanf litourgise swsta
			if(scanfReturn==1){/*ola kala*/}
			else{ printf("error in %d,%d. \n",i,j); return 1;}

			if(fabs(myvar2-arrayYk[i*dimensions+j])>0.5) errors++;
		}
	}
	
	
	printf("The number of errors is %d \n",errors);
	
    

  
  
  return 0;
}